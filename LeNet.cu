#include "hip/hip_runtime.h"
#include "hello_world_cuda.cu"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int main(){
    
    int nin = 32; // dimensions matrice d'entrée
    int nout1 = 28; // dimensions matrice de sortie de la première couche convolutive
    int cout1 = 6; // Nombre de canaux de sortie de la première couche convolutive
    int nmaxpool = 14; // Dimensions de la matrice après max pooling
    int nkernel = 5; //Dimensions du noyau de convolution de la première couche convolutive

// Allocation de la mémoire
    float* raw_data = (float*) malloc(sizeof(float)*nin*nin);
    float* C1_data = (float*) malloc(sizeof(float)*nout1*nout1*cout1);
    float* S1_data = (float*) malloc(sizeof(float)*nmaxpool*nmaxpool*cout1);
    float* C1_kernel = (float*) malloc(sizeof(float)*nkernel*nkernel*cout1);

// Initialisation de la mémoire dans le gpu
    float* raw_data_gpu;
    float* C1_data_gpu;
    float* S1_data_gpu;
    float* C1_kernel_gpu;
    (float*) hipMalloc((void **) &raw_data_gpu, sizeof(float)*nin*nin);
    (float*) hipMalloc((void **) &C1_data_gpu, sizeof(float)*nout1*nout1*cout1);
    (float*) hipMalloc((void **) &S1_data_gpu, sizeof(float)*nmaxpool*nmaxpool*cout1);
    (float*) hipMalloc((void **) &C1_kernel_gpu, sizeof(float)*nkernel*nkernel*cout1);

//Initialisation des matrices
    MatrixInit(raw_data, nin, nin);
    MatrixInitChannel(C1_data, nout1, nout1, cout1);
    MatrixInitChannel(S1_data, nmaxpool, nmaxpool, cout1);
    MatrixInitChannel(C1_kernel, nkernel, nkernel, cout1);

//Copier la mémoire du CPU vers le GPU
    hipMemcpy(raw_data_gpu,raw_data,sizeof(float)*nin*nin,hipMemcpyHostToDevice);
    hipMemcpy(C1_data_gpu,C1_data,sizeof(float)*nout1*nout1*cout1,hipMemcpyHostToDevice);
    hipMemcpy(S1_data_gpu,S1_data,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyHostToDevice);
    hipMemcpy(C1_kernel_gpu,C1_kernel,sizeof(float)*nkernel*nkernel*cout1,hipMemcpyHostToDevice);



return 0;
}