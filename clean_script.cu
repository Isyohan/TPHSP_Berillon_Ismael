#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "functions.cu"

int main(){

    srand( time(NULL) );


    int nin = 32; // dimensions matrice d'entrée
    int cin = 1;

    int nout1 = 28; // dimensions matrice de sortie de la première couche convolutive
    int cout1 = 6; // Nombre de canaux de sortie de la première couche convolutive
    
    int nmaxpool = 14; // Dimensions de la matrice après mean pooling

    int nout2 = 10 // dimensions matrice de sortie de la second couche convolutive
    int cout2 = 16; // Nombre de canaux de sortie de la seconde couche convolutive

    int nmeanpool2 = 5; // Dimensions de la matrice après second mean pooling

    int nkernel = 5; //Dimensions du noyau de convolution de la première & seconde couche convolutive

    int nd_1 = 400; // Dimension du vecteur de la matrice de sortie applatie = 5*5*16
    int nd_2 = 120; // Dimension de sortie de la première couche linéaire
    int nd_3 = 84; // Dimension de sortie de la deuxième couche linéaire
    int nd_4 = 10; // Dimension de sortie de la troisième couche linéaire

// Allocation de la mémoire
    float* raw_data = (float*) malloc(sizeof(float)*nin*nin*cin);

    float* C1_kernel = (float*) malloc(sizeof(float)*nkernel*nkernel*cout1);
    float* biais1 = (float*) malloc(sizeof(float)*cout1);

    float* Mout = (float*) malloc(sizeof(float)*nout1*nout1*cout1);
    float* Moutpool = (float*) malloc(sizeof(float)*nmaxpool*nmaxpool*cout1);

    float* C2_kernel = (float*) malloc(sizeof(float)*nkernel*nkernel*cout1*cout2);
    float* biais2 = (float*) malloc(sizeof(float)*cout2);

    float* Mout2=(float*) malloc(sizeof(float)*nout2*nout2*cout2);
    float* Moutpool2 = (float*) malloc(sizeof(float)*nmeanpool2*nmeanpool2*cout2);


    float* M_dense1=(float*) malloc(sizeof(float)*nd_2*nd_1);
    float* V_out1= (float*) malloc(sizeof(float)*nd_2);
    float* biais3 = (float*) malloc(sizeof(float)*nd_2);

    float* M_dense2=(float*) malloc(sizeof(float)*nd_3*nd_2);
    float* V_out2= (float*) malloc(sizeof(float)*nd_3);
    float* biais4 = (float*) malloc(sizeof(float)*nd_3);

    float* M_dense3=(float*) malloc(sizeof(float)*nd_4*nd_3);
    float* V_out3= (float*) malloc(sizeof(float)*nd_4);
    float* biais5 = (float*) malloc(sizeof(float)*nd_4);
    

// Initialisation de la mémoire dans le gpu


    float* raw_data_gpu; (float*) hipMalloc((void **) &raw_data_gpu, sizeof(float)*nin*nin);

    float* C1_kernel_gpu; (float*) hipMalloc((void **) &C1_kernel_gpu, sizeof(float)*nkernel*nkernel*cout1);
    float* biais1_gpu; (float*) hipMalloc((void **) &biais1_gpu, sizeof(float)*cout1);

    float* Mout_gpu; (float*) hipMalloc((void **) &Mout_gpu, sizeof(float)*nout1*nout1*cout1);
    float* Moutpool_gpu; (float*) hipMalloc((void **) &Moutpool_gpu, sizeof(float)*nmaxpool*nmaxpool*cout1);

    float* C2_kernel_gpu; (float*) hipMalloc((void **) &C2_kernel_gpu, sizeof(float)*nkernel*nkernel*cout1*cout2)
    float* biais2_gpu; (float*) hipMalloc((void **) &biais1_gpu, sizeof(float)*cout2);

    float* Mout2_gpu; (float*) hipMalloc((void **) &Mout2_gpu, sizeof(float)*nout2*nout2*cout2);
    float* Moutpool2_gpu; (float*) hipMalloc((void **) &Moutpool2_gpu, sizeof(float)*nmeanpool2*nmeanpool2*cout2);


    float* M_dense1_gpu; (float*) hipMalloc((void **) &M_dense1_gpu, sizeof(float)*nd_2*nd_1);
    float* V_out1_gpu;
    float* biais3_gpu;

    float* M_dense2_gpu; (float*) hipMalloc((void **) &M_dense2_gpu, sizeof(float)*nd_3*nd_2);
    float* V_out2_gpu;
    float* biais4_gpu;

    float* M_dense3_gpu; (float*) hipMalloc((void **) &M_dense3_gpu, sizeof(float)*nd_4*nd_3);
    float* V_out3_gpu;
    float* biais5_gpu;

    (float*) hipMalloc((void **) &raw_data_gpu, sizeof(float)*nin*nin);
    (float*) hipMalloc((void **) &C1_kernel_gpu, sizeof(float)*nkernel*nkernel*cout1);
    (float*) hipMalloc((void **) &Mout_gpu, sizeof(float)*nout1*nout1*cout1);
    (float*) hipMalloc((void **) &Moutpool_gpu, sizeof(float)*nmaxpool*nmaxpool*cout1);

    (float*) hipMalloc((void **) &biais1_gpu, sizeof(float)*cout1);
    (float*) hipMalloc((void **) &biais2_gpu, sizeof(float)*cout2);
    (float*) hipMalloc((void **) &biais3_gpu, sizeof(float)*nd_2);
    (float*) hipMalloc((void **) &biais4_gpu, sizeof(float)*nd_3);
    (float*) hipMalloc((void **) &biais5_gpu, sizeof(float)*nd_4);

//Initialisation des matrices
    MatrixInit(raw_data, nin, nin);

    MatrixInitChannel(C1_kernel, nkernel, nkernel, cout1);
    //MatrixInitChannel(Mout, nout1, nout1, cout1);
    //MatrixInitChannel(Moutpool,nmaxpool,nmaxpool,cout1);
    MatrixInit(biais1, cout1, 1);

    MatrixInitChannel(C2_kernel, nkernel, nkernel, cout1*cout2);
    MatrixInit(biais2, cout2, 1);






    //biais1[0]=0;

//Copier la mémoire du CPU vers le GPU
    hipMemcpy(raw_data_gpu,raw_data,sizeof(float)*nin*nin,hipMemcpyHostToDevice);
    hipMemcpy(C1_data_gpu,C1_data,sizeof(float)*nout1*nout1*cout1,hipMemcpyHostToDevice);
    hipMemcpy(S1_data_gpu,S1_data,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyHostToDevice);
    hipMemcpy(C1_kernel_gpu,C1_kernel,sizeof(float)*nkernel*nkernel*cout1,hipMemcpyHostToDevice);
    hipMemcpy(Mout_gpu,Mout,sizeof(float)*nout1*nout1*cout1,hipMemcpyHostToDevice);
    hipMemcpy(Moutpool_gpu,Moutpool,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyHostToDevice);
    hipMemcpy(biais1_gpu,biais1,sizeof(float)*cout1,hipMemcpyHostToDevice);

    //Conv2d<<<nout1,nout1>>>(raw_data_gpu, C1_kernel_gpu, Mout_gpu, nin, nkernel, 1, cout1, biais1_gpu); // Convolution
    Conv2d_multi_channel_in<<<nout1,nout1>>>(raw_data_gpu, C1_kernel_gpu, Mout_gpu, nin, nkernel, 1, cout1, biais1_gpu);
    AveragePoolingGlobal<<<nmaxpool,nmaxpool>>>(Mout_gpu, Moutpool_gpu, nmaxpool,2 , cout1); // Maxpooling
    
 
    hipMemcpy(Mout,Mout_gpu,sizeof(float)*nout1*nout1*cout1,hipMemcpyDeviceToHost); // Envoi de la matrice vers le cpu 
    hipMemcpy(Moutpool,Moutpool_gpu,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyDeviceToHost);    


    printf("data : \n");
    MatrixPrint(raw_data,nin,nin);
    printf("\n");

    printf("kernels : \n");
    MatrixPrintChannel(C1_kernel, nkernel, nkernel,cout1);
    printf("\n");

    printf("biais : \n");
    MatrixPrintChannel(biais1, cout1, 1,1);
    printf("\n");

    printf("Convolutioned : \n");
    MatrixPrintChannel(Mout, nout1, nout1,cout1); // Affichage du résultat de la convolution
    printf("\n");

    printf("Maxpooled : \n");
    MatrixPrintChannel(Moutpool, nmaxpool,nmaxpool,cout1); // Affichage du résultat après max pooling

/*
    int n_in = 2;
    int n_out = 1;
    float* V_in= (float*) malloc(sizeof(float)*n_in);
    float* V_out = (float*) malloc(sizeof(float)*n_out);
    float* biais = (float*) malloc(sizeof(float)*n_out);
    float* M_poids = (float*) malloc(sizeof(float)*n_in*n_out);

    float* V_in_gpu;
    float* V_out_gpu;
    float* biais_gpu;
    float* M_poids_gpu;

    (float*) hipMalloc((void **) &V_in_gpu, sizeof(float)*n_in);
    (float*) hipMalloc((void **) &V_out_gpu, sizeof(float)*n_out);
    (float*) hipMalloc((void **) &biais_gpu, sizeof(float)*n_out);
    (float*) hipMalloc((void **) &M_poids_gpu, sizeof(float)*n_in*n_out);

    MatrixInit(V_in, n_in, 1);
    MatrixInit(biais, n_out, 1);
    MatrixInit(M_poids, n_out, n_in);

    hipMemcpy(V_in_gpu,V_in,sizeof(float)*n_in,hipMemcpyHostToDevice);
    hipMemcpy(biais_gpu,biais,sizeof(float)*n_out,hipMemcpyHostToDevice);
    hipMemcpy(M_poids_gpu,M_poids,sizeof(float)*n_in*n_out,hipMemcpyHostToDevice);

    Dense<<<1,n_out>>>(V_in_gpu, V_out_gpu, M_poids_gpu, biais_gpu, n_in, n_out);
    //DenseNormal(V_in, V_out, M_poids, biais, n_in, n_out);
    hipMemcpy(V_out,V_out_gpu,sizeof(float)*n_out,hipMemcpyDeviceToHost);   

    printf("Test Dense : \n");
    printf("V_in : \n");
    MatrixPrint(V_in, n_in, 1);
    printf("biais : \n");
    MatrixPrint(biais, n_out, 1);
    printf("M_poids : \n");
    MatrixPrint(M_poids, n_out, n_in);
    printf("V_out : \n");
    MatrixPrint(V_out, n_out, 1);*/

    hipDeviceSynchronize();


return 0;
}