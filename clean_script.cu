#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "functions.cu"

int main(){

    srand( time(NULL) );


    int nin = 4; // dimensions matrice d'entrée
    int nout1 = 4; // dimensions matrice de sortie de la première couche convolutive
    int cout1 = 1; // Nombre de canaux de sortie de la première couche convolutive
    int nmaxpool = 2; // Dimensions de la matrice après max pooling
    int nkernel = 1; //Dimensions du noyau de convolution de la première couche convolutive

// Allocation de la mémoire
    float* raw_data = (float*) malloc(sizeof(float)*nin*nin);
    float* C1_data = (float*) malloc(sizeof(float)*nout1*nout1*cout1);
    float* S1_data = (float*) malloc(sizeof(float)*nmaxpool*nmaxpool*cout1);
    float* C1_kernel = (float*) malloc(sizeof(float)*nkernel*nkernel*cout1);
    float* Mout = (float*) malloc(sizeof(float)*nout1*nout1*cout1);
    float* Moutpool = (float*) malloc(sizeof(float)*nmaxpool*nmaxpool*cout1);
    
// Initialisation de la mémoire dans le gpu
    float* raw_data_gpu;
    float* C1_data_gpu;
    float* S1_data_gpu;
    float* C1_kernel_gpu;
    float* Mout_gpu;
    float* Moutpool_gpu;
    (float*) hipMalloc((void **) &raw_data_gpu, sizeof(float)*nin*nin);
    (float*) hipMalloc((void **) &C1_data_gpu, sizeof(float)*nout1*nout1*cout1);
    (float*) hipMalloc((void **) &S1_data_gpu, sizeof(float)*nmaxpool*nmaxpool*cout1);
    (float*) hipMalloc((void **) &C1_kernel_gpu, sizeof(float)*nkernel*nkernel*cout1);
    (float*) hipMalloc((void **) &Mout_gpu, sizeof(float)*nout1*nout1*cout1);
    (float*) hipMalloc((void **) &Moutpool_gpu, sizeof(float)*nmaxpool*nmaxpool*cout1);

//Initialisation des matrices
    MatrixInit(raw_data, nin, nin);
    MatrixInitChannel(C1_data, nout1, nout1, cout1);
    MatrixInitChannel(S1_data, nmaxpool, nmaxpool, cout1);
    MatrixInitChannel(C1_kernel, nkernel, nkernel, cout1);
    MatrixInitChannel(Mout, nout1, nout1, cout1);
    MatrixInitChannel(Moutpool,nmaxpool,nmaxpool,cout1);

//Copier la mémoire du CPU vers le GPU
    hipMemcpy(raw_data_gpu,raw_data,sizeof(float)*nin*nin,hipMemcpyHostToDevice);
    hipMemcpy(C1_data_gpu,C1_data,sizeof(float)*nout1*nout1*cout1,hipMemcpyHostToDevice);
    hipMemcpy(S1_data_gpu,S1_data,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyHostToDevice);
    hipMemcpy(C1_kernel_gpu,C1_kernel,sizeof(float)*nkernel*nkernel*cout1,hipMemcpyHostToDevice);
    hipMemcpy(Mout_gpu,Mout,sizeof(float)*nout1*nout1*cout1,hipMemcpyHostToDevice);
    hipMemcpy(Moutpool_gpu,Moutpool,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyHostToDevice);    

    Conv2d<<<nout1,nout1>>>(raw_data_gpu, C1_kernel_gpu, Mout_gpu, nin, nkernel, 1, cout1); // Convolution
    MaxPoolingGlobal<<<nmaxpool,nmaxpool>>>(Mout_gpu, Moutpool_gpu, nmaxpool,2 , cout1); // Maxpooling
    
 
    hipMemcpy(Mout,Mout_gpu,sizeof(float)*nout1*nout1*cout1,hipMemcpyDeviceToHost); // Envoi de la matrice vers le cpu 
    hipMemcpy(Moutpool,Moutpool_gpu,sizeof(float)*nmaxpool*nmaxpool*cout1,hipMemcpyDeviceToHost);    


    printf("data : \n");
    MatrixPrint(raw_data,nin,nin);
    printf("\n");

    printf("kernels : \n");
    MatrixPrintChannel(C1_kernel, nkernel, nkernel,cout1);
    printf("\n");

    printf("Convolutionned : \n");
    MatrixPrintChannel(Mout, nout1, nout1,cout1); // Affichage du résultat de la convolution
    printf("\n");

    printf("Maxpooled : \n");
    MatrixPrintChannel(Moutpool, nmaxpool,nmaxpool,cout1); // Affichage du résultat après max pooling


    hipDeviceSynchronize();


return 0;
}