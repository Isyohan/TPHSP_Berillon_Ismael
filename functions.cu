
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixInitChannel(float *M, int n, int p, int c){
	for(int i =0; i<n*p*c; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrintChannel(float *M,int n,int p ,int c){
    for (int ch=0;ch<c;ch++){
        for (int i=0;i<n;i++){
            for(int j=0;j<p;j++){
                printf("%1.2f\t",M[i*p+j+n*p*ch]);
            }
            printf("\n");
        }
        printf("\n\n");
    }
}




__device__ float MatrixMulTermToTerm(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

float MatrixMulTermToTermNormal(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

__device__ void ChooseChannel(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void ChooseChannelNormal(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void SubMatrixNormal(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}
__device__ void SubMatrixDevice(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}

void ConvNormal(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);
    for (int i=0;i<nout;i++){
        for (int j=0;j<nout;j++){
            SubMatrixNormal(Min,subM,nin,nkernel,channel_in,i,j);
            for (int ch=0 ; ch<channel_kernel ; ch++){
                ChooseChannelNormal(kernels,oneChannelKernel,nkernel,ch);
                Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTermNormal(subM,oneChannelKernel,nkernel);
            }
        }
    }
    
}

__device__ float activation_tanh(float M){
    float tan_h = tanhf(M);
    return tan_h;
}


__global__ void Conv2d(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel, float* biais){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);

    int j = blockIdx.x;
    int i = threadIdx.x;


    SubMatrixDevice(Min,subM,nin,nkernel,channel_in,i,j);
    for (int ch=0 ; ch<channel_kernel ; ch++){
        ChooseChannel(kernels,oneChannelKernel,nkernel,ch);
        Mout[i*nout + j + ch*nout*nout]=biais[ch];
        Mout[i*nout + j + ch*nout*nout]+=activation_tanh(MatrixMulTermToTerm(subM,oneChannelKernel,nkernel));
    }
}

__global__ void Conv2d_multi_channel_in(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_out, float* biais){ // Convolution de 6 dans 16
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);

    int j = blockIdx.x;
    int i = threadIdx.x;


    SubMatrixDevice(Min,subM,nin,nkernel,channel_in,i,j);
    for (int ch=0 ; ch<channel_out ; ch++){
        
        Mout[i*nout + j + ch*nout*nout]=biais[ch];
        for(int chi = 0; chi<channel_in; chi++){
            ChooseChannel(kernels,oneChannelKernel,nkernel,ch*channel_in+chi);
            Mout[i*nout + j + ch*nout*nout]+=MatrixMulTermToTerm(subM,oneChannelKernel,nkernel);
            
        }
        Mout[i*nout + j + ch*nout*nout]=activation_tanh(Mout[i*nout + j + ch*nout*nout]);
    }
}

float MaxMatNormal(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

void maxpoolNormal(float* Min, float* Mout, int nout, int taille_maxpooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling*n_channel);
    float* oneChannelMaxpooling = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling);


    for(int i=0;i<nout;i+=1){
        for(int j=0;j<nout;j+=1){
            SubMatrixNormal(Min,subM,nout*taille_maxpooling,taille_maxpooling,n_channel,i*taille_maxpooling,j*taille_maxpooling);

            for (int ch=0;ch<n_channel;ch++){
                ChooseChannelNormal(subM,oneChannelMaxpooling,taille_maxpooling,ch);

                //printf("(%d,%d,%d)\n",i,j,ch);
                //MatrixPrintChannel(oneChannelMaxpooling,taille_maxpooling,taille_maxpooling,1);

                Mout[j + i*nout + ch*nout*nout]=MaxMatNormal(oneChannelMaxpooling,taille_maxpooling);
            }
        }
    }
}


__device__ float MaxMatDevice(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

__device__ float AverageMatDevice(float *F, int red){
    float moy = 0.0;

    for(int i= 0; i < red*red; i++){
        moy += F[i];
    }
    moy = moy/(red*red);
    return moy;
}

__global__ void AveragePoolingGlobal(float* Min, float* Mout, int nout, int taille_averagepooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_averagepooling*taille_averagepooling*n_channel);
    float* oneChannelAveragepooling = (float*) malloc(sizeof(float)*taille_averagepooling*taille_averagepooling);


    int j = blockIdx.x;
    int i = threadIdx.x;
    SubMatrixDevice(Min,subM,nout*taille_averagepooling,taille_averagepooling,n_channel,i*taille_averagepooling,j*taille_averagepooling);

    for (int ch=0;ch<n_channel;ch++){
        ChooseChannel(subM,oneChannelAveragepooling,taille_averagepooling,ch);

                //printf("(%d,%d,%d)\n",i,j,ch);
                //MatrixPrintChannel(oneChannelMaxpooling,taille_maxpooling,taille_maxpooling,1);

        Mout[j + i*nout + ch*nout*nout]=AverageMatDevice(oneChannelAveragepooling,taille_averagepooling);
    }
       
}

__global__ void MaxPoolingGlobal(float* Min, float* Mout, int nout, int taille_maxpooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling*n_channel);
    float* oneChannelMaxpooling = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling);


    int j = blockIdx.x;
    int i = threadIdx.x;
    SubMatrixDevice(Min,subM,nout*taille_maxpooling,taille_maxpooling,n_channel,i*taille_maxpooling,j*taille_maxpooling);

    for (int ch=0;ch<n_channel;ch++){
        ChooseChannel(subM,oneChannelMaxpooling,taille_maxpooling,ch);

                //printf("(%d,%d,%d)\n",i,j,ch);
                //MatrixPrintChannel(oneChannelMaxpooling,taille_maxpooling,taille_maxpooling,1);

        Mout[j + i*nout + ch*nout*nout]=MaxMatDevice(oneChannelMaxpooling,taille_maxpooling);
    }
       
}

void DenseNormal(float* V_in, float* V_out, float* M_poids, float* biais, int n_in, int n_out){
    for(int i = 0; i<n_out;i++){
        V_out[i] = biais[i];
        for(int j = 0; j<n_in;j++){
            V_out[i] += M_poids[i*n_in+j]*V_in[j];
        }
        
    }
}

__global__ void Dense(float* V_in, float* V_out, float* M_poids, float* biais, int n_in, int n_out){

    int i = blockIdx.x;

    
    V_out[i] = biais[i];
    for(int j = 0; j<n_in;j++){
        V_out[i] += M_poids[i*n_in+j]*V_in[j];
    }
    V_out[i] = activation_tanh(V_out[i]);
    
}