
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void helloworld_cuda(){
    printf("Hello\n");
}

void MatrixMul(int *in1,int *in2 ,int a ,int *out ){
    for (int i=0;i<a*a;i++){
        int s=0;
        for (int j=0;j<a;j++){
            s+=in1[j+((int)i/a)*a]*in2[j*a+(i%a)];
             
        }
        out[i]=s;
    }
}

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for(int i = 0; i<n*p;i++){
        Mout[i] = M1[i] + M2[i];
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    int i = blockIdx.x;
    int j = threadIdx.x;
//    printf("%d \t %d \n", i, j);

//    if(i*p+j>=n*p){printf("error");}
    Mout[i*p+j] = M1[i*p+j] + M2[i*p+j];
//    printf("%1.2f \n", Mout[i*p+j]);
}


int main(){
    helloworld_cuda<<<1,1>>>();
    srand( time(NULL) );
/*    int a=3;
    int in1[a*a];
    int in2[a*a];
    int out[a*a];
    for (int i=0; i<a*a;i++){
        in1[i]=i;
        in2[i]=i;
    }
    
    MatrixMul(in1,in2,a,out);
    for (int i=0;i<a*a;i++){
        printf("%d\n",out[i]);
    }*/
    int n = 3; // lignes
    int p = 2; // colonnes

    float* M1 = (float*) malloc(sizeof(float)*n*p);
    float* M2 = (float*) malloc(sizeof(float)*n*p);
    float* Mout = (float*) malloc(sizeof(float)*n*p);

    float* M1gpu;
    float* M2gpu;
    float* Moutgpu;
    (float*) hipMalloc((void **) &M1gpu, sizeof(float)*n*p);
    (float*) hipMalloc((void **) &M2gpu, sizeof(float)*n*p);
    (float*) hipMalloc((void **) &Moutgpu, sizeof(float)*n*p);

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);
    hipMemcpy(M1gpu,M1,sizeof(float)*n*p,hipMemcpyHostToDevice);
    hipMemcpy(M2gpu,M2,sizeof(float)*n*p,hipMemcpyHostToDevice);
//    cudaMemcpy(Moutgpu,Mout,n*p,cudaMemcpyHostToDevice);
    cudaMatrixAdd<<<n,p>>>(M1gpu,M2gpu,Moutgpu,n,p);

 //   cudaMemcpy(M1,M1gpu,n*p,cudaMemcpyDeviceToHost);
 //   cudaMemcpy(M2,M2gpu,n*p,cudaMemcpyDeviceToHost);
    hipMemcpy(Mout,Moutgpu,sizeof(float)*n*p,hipMemcpyDeviceToHost);

    MatrixPrint(M1, n, p);
    MatrixPrint(M2, n, p);
    MatrixPrint(Mout, n, p);
    hipDeviceSynchronize();
    printf("cuda\n");
    return 0;
}

