
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloworld_cuda(){
    printf("Hello\n");
}

void mulmat(int *in1,int *in2 ,int a ,int *out ){
    for (int i=0;i<a*a;i++){
        int s=0;
        for (int j=0;j<a;j++){
            s+=in1[j+((int)i/a)*a]*in2[j*a+(i%a)];
             
        }
        out[i]=s;
    }
}

int main(){
    helloworld_cuda<<<1,1>>>();
    int a=3;
    int in1[a*a];
    int in2[a*a];
    int out[a*a];
    for (int i=0; i<a*a;i++){
        in1[i]=i;
        in2[i]=i;
    }
    
    mulmat(in1,in2,a,out);
    for (int i=0;i<a*a;i++){
        printf("%d\n",out[i]);
    }

    hipDeviceSynchronize();
    printf("cuda\n");
    return 0;
}

