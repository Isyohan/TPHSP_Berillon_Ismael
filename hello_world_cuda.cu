
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void helloworld_cuda(){
    printf("Hello\n");
}

void mulmat(int *in1,int *in2 ,int a ,int *out ){
    for (int i=0;i<a*a;i++){
        int s=0;
        for (int j=0;j<a;j++){
            s+=in1[j+((int)i/a)*a]*in2[j*a+(i%a)];
             
        }
        out[i]=s;
    }
}

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for(int i = 0; i<n*p;i++){
        Mout[i] = M1[i] + M2[i];
    }
}

int main(){
    helloworld_cuda<<<1,1>>>();
    srand( time(NULL) );
/*    int a=3;
    int in1[a*a];
    int in2[a*a];
    int out[a*a];
    for (int i=0; i<a*a;i++){
        in1[i]=i;
        in2[i]=i;
    }
    
    mulmat(in1,in2,a,out);
    for (int i=0;i<a*a;i++){
        printf("%d\n",out[i]);
    }*/
    int n = 3; // lignes
    int p = 2; // colonnes
    float* M1 = (float*) malloc(sizeof(float)*n*p);
    float* M2 = (float*) malloc(sizeof(float)*n*p);
    float* Mout = (float*) malloc(sizeof(float)*n*p);

    MatrixInit(M1, n, p);
    MatrixInit(M2, n, p);
    MatrixAdd(M1,M2,Mout,n,p);
    MatrixPrint(M1, n, p);
    MatrixPrint(M2, n, p);
    MatrixPrint(Mout, n, p);
    hipDeviceSynchronize();
    printf("cuda\n");
    return 0;
}

