
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

__global__ void helloworld_cuda(){
    printf("Hello\n");
}

void MatrixMul(float *in1,float *in2 ,int a ,float *out ){
    /*
    Matrice carrée d'ordre a
    On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d 

    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b 
    c d

    e f
    g h

    Résultat :
    a*e+b*g a*f+b*h
    c*e+d*g c*f+d*h
    
    Indice i désigne les lignes
    Indice j déqigne les colonnes
    */
    for (int i=0;i<a*a;i++){
        int s=0;
        for (int j=0;j<a;j++){
            s+=in1[j+((int)i/a)*a]*in2[j*a+(i%a)];
             
        }
        out[i]=s;
    }
}

__global__ void cudaMatrixMul(float *M1, float*M2, float *Mout, int n){
    int i = blockIdx.x;
    int j = threadIdx.x;
    float temp = 0;

    for(int k = 0; k<n; k++){
        temp+= M1[k+i*n]*M2[n*k+j]; //i : lignes, j : colonnes
    }
    Mout[i*n+j]=temp;
}

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for(int i = 0; i<n*p;i++){
        Mout[i] = M1[i] + M2[i];
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    int i = blockIdx.x;
    int j = threadIdx.x;
//    printf("%d \t %d \n", i, j);

//    if(i*p+j>=n*p){printf("error");}
    Mout[i*p+j] = M1[i*p+j] + M2[i*p+j];
//    printf("%1.2f \n", Mout[i*p+j]);
}


int main(){
    
    srand( time(NULL) );

/*    int a=3;
    int in1[a*a];
    int in2[a*a];
    int out[a*a];
    for (int i=0; i<a*a;i++){
        in1[i]=i;
        in2[i]=i;
    }
    
    MatrixMul(in1,in2,a,out);
    for (int i=0;i<a*a;i++){
        printf("%d\n",out[i]);
    }*/
    int n = 2'000; // lignes
 //   int p = 2; // colonnes

    float* M1 = (float*) malloc(sizeof(float)*n*n);
    float* M2 = (float*) malloc(sizeof(float)*n*n);
    float* Mout = (float*) malloc(sizeof(float)*n*n);

    float* M1gpu;
    float* M2gpu;
    float* Moutgpu;
    (float*) hipMalloc((void **) &M1gpu, sizeof(float)*n*n);
    (float*) hipMalloc((void **) &M2gpu, sizeof(float)*n*n);
    (float*) hipMalloc((void **) &Moutgpu, sizeof(float)*n*n);

    MatrixInit(M1, n, n);
    MatrixInit(M2, n, n);

    //MatrixMul(M1,M2,n,Mout);

    
    hipMemcpy(M1gpu,M1,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(M2gpu,M2,sizeof(float)*n*n,hipMemcpyHostToDevice);

    cudaMatrixMul<<<n,n>>>(M1gpu,M2gpu,Moutgpu,n);

    hipMemcpy(Mout,Moutgpu,sizeof(float)*n*n,hipMemcpyDeviceToHost);
    
    

    //MatrixPrint(M1, n, n);
    //MatrixPrint(M2, n, n);
    //MatrixPrint(Mout, n, n);
    hipDeviceSynchronize();
    printf("n = %d\n",n);
    return 0;
}

