
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

__global__ void helloworld_cuda(){
    printf("Hello\n");
}

void MatrixMul(float *in1,float *in2 ,int a ,float *out ){
    /*
    Matrice carrée d'ordre a
    On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d 

    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b 
    c d

    e f
    g h

    Résultat :
    a*e+b*g a*f+b*h
    c*e+d*g c*f+d*h
    
    Indice i désigne les lignes
    Indice j déqigne les colonnes
    */
    for (int i=0;i<a*a;i++){
        int s=0;
        for (int j=0;j<a;j++){
            s+=in1[j+((int)i/a)*a]*in2[j*a+(i%a)];
             
        }
        out[i]=s;
    }
}

__global__ void cudaMatrixMul(float *M1, float*M2, float *Mout, int n){
    int i = blockIdx.x;
    int j = threadIdx.x;
    float temp = 0;

    for(int k = 0; k<n; k++){
        temp+= M1[k+i*n]*M2[n*k+j]; //i : lignes, j : colonnes
    }
    Mout[i*n+j]=temp;
}

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixInitChannel(float *M, int n, int p, int c){
	for(int i =0; i<n*p*c; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    for(int i = 0; i<n*p;i++){
        Mout[i] = M1[i] + M2[i];
    }
}

__device__ float MatrixMulTermToTerm(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

__device__ void SubMatrix(float *M1, float *Mout, int n, int i, int j,int c,int N){ // Récupérer la matrice de taille 5*5 à partir de l'indice (i,j)

    for(int k = 0; k<n; k++){
        for(int l = 0; l < n; l++){
            Mout[l+k*n] = M1[l+j+(k+i)*N+c*N*N];
        }
    }
}

__device__ void ChooseChannel(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

__global__ void cudaConvolutionMatrix(float *M1, float *M2, float *Mout, int n, int k, int c){ // Réalisation de la convolution
    int i = blockIdx.x;
    int j = threadIdx.x;
    float* M = (float*) malloc(sizeof(float)*k*k); // Sous matrice locale pour la convolution
    float* F = (float*) malloc(sizeof(float)*k*k); // Sous matrice pour chaque canal d'entrée

    //SubMatrix(M1, M, k, i, j,n+k-1);
    for(int ch = 0; ch < c; ch++){ // Pour chaque canal
        SubMatrix(M1, M, k, i, j,ch,n+k-1);
        ChooseChannel(M2, F, k, ch);
        Mout[i*n+j+ch*n*n] = MatrixMulTermToTerm(F,M,k); // Convolution pour chaque canal
    }

}

__device__ float MaxMat(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

__global__ void cudaMaxPooling(float *M1, float *Mout, int red, int nout, int c){
    int i = blockIdx.x;
    int j = threadIdx.x;
    printf("i=%d , j=%d\n",i,j);
    int nin = nout*red;
    float* F = (float*) malloc(sizeof(float)*red*red); // Sous matrice pour chaque canal dans laquelle on va choisir le maximum
    float* SubM = (float*) malloc(sizeof(float)*red*red*c); // Sous matrice de taille 2*2*6

    //SubMatrix(M1, SubM, red, red*i, red*j, nin); // red désigne le paramètre par lequel on va réduire la matrice, ici red = 2

    for(int ch = 0; ch < c; ch++){
        SubMatrix(M1, SubM, red, (red+1)*i, (red+1)*j, ch, nin); // red désigne le paramètre par lequel on va réduire la matrice, ici red = 2
        ChooseChannel(SubM, F, red, ch);
        Mout[i*nout+j+c*nout*nout] =  MaxMat(F,red); // On choisit le maximum de la matrice F 
        Mout[0]=MaxMat(F,red);
    }
}

__global__ void cudaMatrixAdd(float *M1, float *M2, float *Mout, int n, int p){
    int i = blockIdx.x;
    int j = threadIdx.x;
//    printf("%d \t %d \n", i, j);

//    if(i*p+j>=n*p){printf("error");}
    Mout[i*p+j] = M1[i*p+j] + M2[i*p+j];
//    printf("%1.2f \n", Mout[i*p+j]);
}


int mainTP1(){
    
    srand( time(NULL) );

/*    int a=3;
    int in1[a*a];
    int in2[a*a];
    int out[a*a];
    for (int i=0; i<a*a;i++){
        in1[i]=i;
        in2[i]=i;
    }
    
    MatrixMul(in1,in2,a,out);
    for (int i=0;i<a*a;i++){
        printf("%d\n",out[i]);
    }*/
    int n = 2'000; // lignes
 //   int p = 2; // colonnes

    float* M1 = (float*) malloc(sizeof(float)*n*n);
    float* M2 = (float*) malloc(sizeof(float)*n*n);
    float* Mout = (float*) malloc(sizeof(float)*n*n);

    float* M1gpu;
    float* M2gpu;
    float* Moutgpu;
    (float*) hipMalloc((void **) &M1gpu, sizeof(float)*n*n);
    (float*) hipMalloc((void **) &M2gpu, sizeof(float)*n*n);
    (float*) hipMalloc((void **) &Moutgpu, sizeof(float)*n*n);

    MatrixInit(M1, n, n);
    MatrixInit(M2, n, n);

    //MatrixMul(M1,M2,n,Mout);

    
    hipMemcpy(M1gpu,M1,sizeof(float)*n*n,hipMemcpyHostToDevice);
    hipMemcpy(M2gpu,M2,sizeof(float)*n*n,hipMemcpyHostToDevice);

    cudaMatrixMul<<<n,n>>>(M1gpu,M2gpu,Moutgpu,n);

    hipMemcpy(Mout,Moutgpu,sizeof(float)*n*n,hipMemcpyDeviceToHost);
    
    

    //MatrixPrint(M1, n, n);
    //MatrixPrint(M2, n, n);
    //MatrixPrint(Mout, n, n);
    hipDeviceSynchronize();
    printf("n = %d\n",n);
    return 0;
}

