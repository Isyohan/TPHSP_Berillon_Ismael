
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixInitChannel(float *M, int n, int p, int c){
	for(int i =0; i<n*p*c; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrintChannel(float *M,int n,int p ,int c){
    for (int ch=0;ch<c;ch++){
        for (int i=0;i<n;i++){
            for(int j=0;j<p;j++){
                printf("%1.2f\t",M[i*p+j+n*p*ch]);
            }
            printf("\n");
        }
        printf("\n\n");
    }
}




__device__ float MatrixMulTermToTerm(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

float MatrixMulTermToTermNormal(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

__device__ void ChooseChannel(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void ChooseChannelNormal(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void SubMatrixNormal(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}
__device__ void SubMatrixDevice(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}

void ConvNormal(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);
    for (int i=0;i<nout;i++){
        for (int j=0;j<nout;j++){
            SubMatrixNormal(Min,subM,nin,nkernel,channel_in,i,j);
            for (int ch=0 ; ch<channel_kernel ; ch++){
                ChooseChannelNormal(kernels,oneChannelKernel,nkernel,ch);
                Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTermNormal(subM,oneChannelKernel,nkernel);
            }
        }
    }
    
}

__global__ void Conv2d(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);

    int j = blockIdx.x;
    int i = threadIdx.x;


    SubMatrixDevice(Min,subM,nin,nkernel,channel_in,i,j);
    for (int ch=0 ; ch<channel_kernel ; ch++){
        ChooseChannel(kernels,oneChannelKernel,nkernel,ch);
        Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTerm(subM,oneChannelKernel,nkernel);
    }
}


float MaxMatNormal(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

void maxpoolNormal(float* Min, float* Mout, int nout, int taille_maxpooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling*n_channel);
    float* oneChannelMaxpooling = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling);


    for(int i=0;i<=nout;i+=1){
        for(int j=0;j<=nout;j+=1){
            SubMatrixNormal(Min,subM,nout*taille_maxpooling,taille_maxpooling,n_channel,i*taille_maxpooling,j*taille_maxpooling);

            for (int ch=0;ch<n_channel;ch++){
                ChooseChannelNormal(subM,oneChannelMaxpooling,taille_maxpooling,ch);

                //printf("(%d,%d,%d)\n",i,j,ch);
                //MatrixPrintChannel(oneChannelMaxpooling,taille_maxpooling,taille_maxpooling,1);

                Mout[j + i*nout + ch*nout*nout]=MaxMatNormal(oneChannelMaxpooling,taille_maxpooling);
            }
        }
    }
}


__device__ float MaxMatDevice(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

__global__ void MaxPoolingGlobal(float* Min, float* Mout, int nout, int taille_maxpooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling*n_channel);
    float* oneChannelMaxpooling = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling);


    int j = blockIdx.x;
    int i = threadIdx.x;

    SubMatrixDevice(Min,subM,nout*taille_maxpooling,taille_maxpooling,n_channel,i*taille_maxpooling,j*taille_maxpooling);

    for (int ch=0;ch<n_channel;ch++){
        ChooseChannel(subM,oneChannelMaxpooling,taille_maxpooling,ch);

                //printf("(%d,%d,%d)\n",i,j,ch);
                //MatrixPrintChannel(oneChannelMaxpooling,taille_maxpooling,taille_maxpooling,1);

        Mout[j + i*nout + ch*nout*nout]=MaxMatDevice(oneChannelMaxpooling,taille_maxpooling);
    }
       
}
