
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixInitChannel(float *M, int n, int p, int c){
	for(int i =0; i<n*p*c; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrintChannel(float *M,int n,int p ,int c){
    for (int ch=0;ch<c;ch++){
        for (int i=0;i<n;i++){
            for(int j=0;j<p;j++){
                printf("%1.2f\t",M[i*p+j+n*p*ch]);
            }
            printf("\n");
        }
        printf("\n\n");
    }
}




__device__ float MatrixMulTermToTerm(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

float MatrixMulTermToTermNormal(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

__device__ void SubMatrix(float *M1, float *Mout, int n, int i, int j){ // Récupérer la matrice de taille 5*5 à partir de l'indice (i,j)
    int N = 32;
    for(int k = 0; k<n; k++){
        for(int l = 0; l < n; l++){
            Mout[l+k*n] = M1[l+j+(k+i)*N];
        }
    }
}

__device__ void ChooseChannel(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void ChooseChannelNormal(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void SubMatrixNormal(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}
__device__ void SubMatrixDevice(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}


__global__ void cudaConvolutionMatrix(float *M1, float *M2, float *Mout, int n, int k, int c1,int c2){ // Réalisation de la convolution
    int i = blockIdx.x;
    int j = threadIdx.x;
    float* M = (float*) malloc(sizeof(float)*k*k);
    float* F = (float*) malloc(sizeof(float)*k*k);

    SubMatrixDevice(M1, M, n, k, c1, i, j);
    for(int ch = 0; ch < c2; ch++){ // Pour chaque canal
        ChooseChannel(M2, F, k, ch);
        Mout[i*n+j+ch*n*n] = MatrixMulTermToTerm(F,M,k); // Convolution pour chaque canal
    }

}

void ConvNormal(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);
    for (int i=0;i<nout;i++){
        for (int j=0;j<nout;j++){
            SubMatrixNormal(Min,subM,nin,nkernel,channel_in,i,j);
            for (int ch=0 ; ch<channel_kernel ; ch++){
                ChooseChannelNormal(kernels,oneChannelKernel,nkernel,ch);
                Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTermNormal(subM,oneChannelKernel,nkernel);
            }
        }
    }
    
}

__global__ void Conv2d(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);

    int j = blockIdx.x;
    int i = threadIdx.x;


    SubMatrixDevice(Min,subM,nin,nkernel,channel_in,i,j);
    for (int ch=0 ; ch<channel_kernel ; ch++){
        ChooseChannel(kernels,oneChannelKernel,nkernel,ch);
        Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTerm(subM,oneChannelKernel,nkernel);
    }
}
