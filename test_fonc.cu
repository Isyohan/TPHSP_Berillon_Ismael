
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

void MatrixInit(float *M, int n, int p){
	for(int i =0; i<n*p; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixInitChannel(float *M, int n, int p, int c){
	for(int i =0; i<n*p*c; i++){
		M[i] = (float)2*((float)rand()-((float)RAND_MAX)/2)/(float)RAND_MAX;
	} // [rand - (RAND_MAX/2)]/ (RAND_MAX/2) pour avoir un nombre aléatoire entre -1 et 1
}

void MatrixPrint(float *M, int n, int p){
    /* On stocke la matrice dans une liste de dimension 1 et de taille n*p:

    a b c d e f
    
    Cette liste désigne la matrice de taille n lignes et p colonnes :
    
    a b c 
    d e f

    L'indice i désigne les lignes
    L'indice j désigne les colonnes
    */
    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            printf("%1.2f\t",M[i*p+j]);
        }
        printf("\n");
    }
    printf("\n");
}

void MatrixPrintChannel(float *M,int n,int p ,int c){
    for (int ch=0;ch<c;ch++){
        for (int i=0;i<n;i++){
            for(int j=0;j<p;j++){
                printf("%1.2f\t",M[i*p+j+n*p*ch]);
            }
            printf("\n");
        }
        printf("\n\n");
    }
}




__device__ float MatrixMulTermToTerm(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

float MatrixMulTermToTermNormal(float *M1, float *M2, int n){ // Pour faire la convolution
    float sum = 0;
    for(int i = 0; i<n*n;i++){
        sum += M1[i]*M2[i];
    }
    return sum;
}

__device__ void ChooseChannel(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void ChooseChannelNormal(float *M1, float *Mout, int n, int c){
    for(int i=0; i<n*n; i++){
        Mout[i] = M1[i+c*n*n];
    }

}

void SubMatrixNormal(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}
__device__ void SubMatrixDevice(float *M1, float *Mout,int nin, int n,int c, int i, int j){ // Récupérer la matrice de taille n*n à partir de l'indice (i,j)
    for (int ch=0;ch<c;ch++){
        for(int k = 0; k<n; k++){
            for(int l = 0; l < n; l++){
                Mout[k + l*n + n*n*ch] = M1[k+j + (l+i)*nin + nin*nin*ch];
            }
        }
    }
}

void ConvNormal(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);
    for (int i=0;i<nout;i++){
        for (int j=0;j<nout;j++){
            SubMatrixNormal(Min,subM,nin,nkernel,channel_in,i,j);
            for (int ch=0 ; ch<channel_kernel ; ch++){
                ChooseChannelNormal(kernels,oneChannelKernel,nkernel,ch);
                Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTermNormal(subM,oneChannelKernel,nkernel);
            }
        }
    }
    
}

__global__ void Conv2d(float* Min ,float* kernels ,float* Mout ,int nin ,int nkernel ,int channel_in ,int channel_kernel){
    int nout=nin-nkernel+1;
    float* subM = (float*) malloc(sizeof(float)*nkernel*nkernel);
    float* oneChannelKernel = (float*) malloc(sizeof(float)*nkernel*nkernel);

    int j = blockIdx.x;
    int i = threadIdx.x;


    SubMatrixDevice(Min,subM,nin,nkernel,channel_in,i,j);
    for (int ch=0 ; ch<channel_kernel ; ch++){
        ChooseChannel(kernels,oneChannelKernel,nkernel,ch);
        Mout[i*nout + j + ch*nout*nout]=MatrixMulTermToTerm(subM,oneChannelKernel,nkernel);
    }
}


float MaxMatNormal(float *F, int red){
    float max = -1.0;

    for(int i = 0; i < red*red; i++){
        if(max<F[i]){
            max = F[i];
        }
    }
    return max;
}

void maxpool(float* Min, float* Mout, int nout, int taille_maxpooling, int n_channel){
    float* subM = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling*n_channel);
    float* oneChannelMaxpooling = (float*) malloc(sizeof(float)*taille_maxpooling*taille_maxpooling);


    for(int i=0;i<nout;i+=taille_maxpooling){
        for(int j=0;j<nout;j+=taille_maxpooling){
            SubMatrixNormal(Min,subM,nout*taille_maxpooling,taille_maxpooling,n_channel,i,j);
            
            for (int ch=0;ch<n_channel;ch++){
                ChooseChannelNormal(subM,oneChannelMaxpooling,taille_maxpooling,ch);
                Mout[j + i*nout + ch*nout*nout]=MaxMatNormal(oneChannelMaxpooling,taille_maxpooling);
            }
        }
    }
}
/*
void cudaMaxPoolingNormal(float *M1, float *Mout, int red, int nout, int c){
    int i = blockIdx.x;
    int j = threadIdx.x;
    printf("i=%d , j=%d\n",i,j);
    int nin = nout*red;
    float* F = (float*) malloc(sizeof(float)*red*red); // Sous matrice pour chaque canal dans laquelle on va choisir le maximum
    float* SubM = (float*) malloc(sizeof(float)*red*red*c); // Sous matrice de taille 2*2*6

    SubMatrix(M1, SubM, red, red*i, red*j, nin); // red désigne le paramètre par lequel on va réduire la matrice, ici red = 2

    for(int ch = 0; ch < c; ch++){
        SubMatrixNormal(M1, SubM, red, (red+1)*i, (red+1)*j, ch, nin); // red désigne le paramètre par lequel on va réduire la matrice, ici red = 2
        ChooseChannel(SubM, F, red, ch);
        Mout[i*nout+j+c*nout*nout] =  MaxMat(F,red); // On choisit le maximum de la matrice F 
        Mout[0]=MaxMat(F,red);
    }
}
*/
