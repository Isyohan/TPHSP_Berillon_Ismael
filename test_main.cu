#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "test_fonc.cu"


int main(){
    srand( time(NULL) );

    int nin=4;
    int nb_channel_in=1;
    float* data = (float*) malloc(sizeof(float)*nin*nin*nb_channel_in);
    float* data_gpu;
    (float*) hipMalloc((void **) &data_gpu, sizeof(float)*nin*nin*nb_channel_in);

    MatrixInitChannel(data, nin, nin,nb_channel_in);

    hipMemcpy(data_gpu,data,sizeof(float)*nin*nin*nb_channel_in,hipMemcpyHostToDevice);


    
    int nkernel=1;
    int ch_kernel=2;
    float* kernel = (float*) malloc(sizeof(float)*nkernel*nkernel*ch_kernel);
    float* kernel_gpu;
    (float*) hipMalloc((void **) &kernel_gpu, sizeof(float)*nkernel*nkernel*ch_kernel);
    
    MatrixInitChannel(kernel, nkernel, nkernel,ch_kernel);
 

    hipMemcpy(kernel_gpu,kernel,sizeof(float)*nkernel*nkernel*ch_kernel,hipMemcpyHostToDevice);



    printf("kernel : \n");
    MatrixPrintChannel(kernel,nkernel,nkernel,ch_kernel);
    
    printf("data : \n");
    MatrixPrintChannel(data,nin,nin,nb_channel_in);


    int nout=nin-nkernel+1;
    
    float* out = (float*) malloc(sizeof(float)*nout*nout*ch_kernel);
    float* out_gpu;
    (float*) hipMalloc((void **) &out_gpu, sizeof(float)*nout*nout*ch_kernel);




//    ConvNormal(data,kernel,out,nin,nkernel,nb_channel_in,ch_kernel);
    Conv2d<<<nout,nout>>>(data_gpu,kernel_gpu,out_gpu,nin,nkernel,nb_channel_in,ch_kernel);

    hipMemcpy(out,out_gpu,sizeof(float)*nout*nout*ch_kernel,hipMemcpyDeviceToHost); // Envoi de la matrice vers le cpu 

    printf("out:\n");
    MatrixPrintChannel(out,nout,nout,ch_kernel);

    int taille_maxpooling=2;
    int nmaxpool=nout/taille_maxpooling;

    float* outmaxpool = (float*) malloc(sizeof(float)*nmaxpool*nmaxpool*ch_kernel);
    float* outmaxpool_gpu;
    (float*) hipMalloc((void **) &outmaxpool_gpu, sizeof(float)*nmaxpool*nmaxpool*ch_kernel);
    
    
    MaxPoolingGlobal<<<nmaxpool,nmaxpool>>>(out_gpu,outmaxpool_gpu,nmaxpool,taille_maxpooling,ch_kernel);

    hipMemcpy(outmaxpool,outmaxpool_gpu,sizeof(float)*nmaxpool*nmaxpool*ch_kernel,hipMemcpyDeviceToHost); // Envoi de la matrice vers le cpu 

    hipDeviceSynchronize();

    printf("out maxpool :\n");
    MatrixPrintChannel(outmaxpool,nmaxpool,nmaxpool,ch_kernel);

    return 0;
}